#include "hip/hip_runtime.h"
//  Project AC-SpGEMM
//  https://www.tugraz.at/institute/icg/research/team-steinberger/
//
//  Copyright (C) 2018 Institute for Computer Graphics and Vision,
//                     Graz University of Technology
//
//  Author(s):  Martin Winter - martin.winter (at) icg.tugraz.at
//              Daniel Mlakar - daniel.mlakar (at) icg.tugraz.at
//              Rhaleb Zayer - rzayer (at) mpi-inf.mpg.de
//              Hans-Peter Seidel - hpseidel (at) mpi-inf.mpg.de
//              Markus Steinberger - steinberger ( at ) icg.tugraz.at
//
//  Permission is hereby granted, free of charge, to any person obtaining a copy
//  of this software and associated documentation files (the "Software"), to deal
//  in the Software without restriction, including without limitation the rights
//  to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
//  copies of the Software, and to permit persons to whom the Software is
//  furnished to do so, subject to the following conditions:
//
//  The above copyright notice and this permission notice shall be included in
//  all copies or substantial portions of the Software.
//
//  THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
//  IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
//  FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
//  AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
//  LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
//  OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
//  THE SOFTWARE.
//

/*!/------------------------------------------------------------------------------
* Compare.cu
*
* ac-SpGEMM
*
* Authors: Daniel Mlakar, Markus Steinberger, Martin Winter
*------------------------------------------------------------------------------
*/

// Global includes
#include <stdio.h>
#include <stdint.h>

// Local includes
#include "acspgemm/Compare.h"
#include "acspgemm/common.h"

//#define VERIFICATION_TEXT

template <typename IndexType, typename DataType>
__global__ void d_compare(
    int in_rows, 
    int in_cols, 
    const IndexType* __restrict reference_offset, 
    const IndexType* __restrict reference_indices, 
    const DataType* __restrict reference_values,
    const IndexType* __restrict compare_offset, 
    const IndexType* __restrict compare_indices, 
    const DataType* __restrict compare_values, 
    bool compare_data, 
    double epsilon, uint32_t* verification)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if (tid >= in_rows)
        return;

    IndexType ref_offset = reference_offset[tid];
    IndexType comp_offset = compare_offset[tid];
    IndexType ref_number_entries = reference_offset[tid + 1] - ref_offset;
    IndexType comp_number_entries = compare_offset[tid + 1] - comp_offset;

	if (ref_number_entries != comp_number_entries)
	{
#ifdef VERIFICATION_TEXT
		printf("---------- Row: %u | Row length not identical: (Ref|Comp) : (%u|%u)\n",tid, ref_number_entries, comp_number_entries);
#endif
		*verification = 1;
	}

	IndexType num_entries = min(ref_number_entries, comp_number_entries);

	for (IndexType i = 0; i < num_entries; ++i)
	{
		if (reference_indices[ref_offset + i] != compare_indices[comp_offset + i])
		{
#ifdef VERIFICATION_TEXT
			printf("Row: %u | Row indices do NOT match: (Ref|Comp) : (%u|%u) - pos: %u/%u\n", tid, reference_indices[ref_offset + i], compare_indices[comp_offset + i], i, num_entries);
#endif
			*verification = 1;
		}
		if (compare_data)
		{
			if (reference_values[ref_offset + i] != compare_values[comp_offset + i])
			{
#ifdef VERIFICATION_TEXT
				printf("Row: %u | Values do NOT match: (Ref|Comp) : (%f|%f) - pos: %u/%u\n", tid, reference_values[ref_offset + i], compare_values[comp_offset + i], i, num_entries);
#endif
				*verification = 1;
			}
		}
	}

	return;
}

namespace ACSpGEMM {

    template <typename IndexType, typename DataType>
    bool Compare(
        const spformat::dCSR<IndexType,DataType>& reference_mat, 
        const spformat::dCSR<IndexType,DataType>& compare_mat, 
        bool compare_data)
    {
        int blockSize(256);
        int gridSize(divup<int>(reference_mat.nrows_ + 1, blockSize));
        double epsilon = 0.1;
        uint32_t* verification, h_verification;
        hipMalloc(&verification, sizeof(uint32_t));
        hipMemset(verification, 0, sizeof(uint32_t));

        d_compare<IndexType, DataType> <<<gridSize, blockSize >>> (reference_mat.nrows_, reference_mat.ncols_,
            reference_mat.row_offsets, reference_mat.col_ids, reference_mat.data,
            compare_mat.row_offsets, compare_mat.col_ids, compare_mat.data,
            compare_data, epsilon, verification);
            
        hipMemcpy(&h_verification, verification, sizeof(uint32_t), hipMemcpyDeviceToHost);
        return (h_verification == 0);
    }

	template bool Compare<uint32_t,float>(const spformat::dCSR<uint32_t,float>& reference_mat, const spformat::dCSR<uint32_t,float>& compare_mat, bool compare_data);
	template bool Compare<uint32_t, double>(const spformat::dCSR<uint32_t,double>& reference_mat, const spformat::dCSR<uint32_t,double>& compare_mat, bool compare_data);
}