/*!/------------------------------------------------------------------------------
 * cuSparseMultiply.cu
 *
 * ac-SpGEMM
 *
 * Authors: Daniel Mlakar, Markus Steinberger, Martin Winter
 *------------------------------------------------------------------------------
 */

#include "cusparse12/include/cuSparseMultiply.h"
#include <hip/hip_runtime.h>

#define CHECK_CUDA(func)                                                       \
  {                                                                            \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
      printf("CUDA API failed at line %d with error: %s (%d)\n", __LINE__,     \
             hipGetErrorString(status), status);                              \
      return EXIT_FAILURE;                                                     \
    }                                                                          \
  }

#define CHECK_CUSPARSE(func)                                                   \
  {                                                                            \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
      printf("CUSPARSE API failed at line %d with error: %s (%d)\n", __LINE__, \
             hipsparseGetErrorString(status), status);                          \
      return EXIT_FAILURE;                                                     \
    }                                                                          \
  }

#define CHECK_CUSPARSE_NORET(func)                                             \
  {                                                                            \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
      printf("CUSPARSE API failed at line %d with error: %s (%d)\n", __LINE__, \
             hipsparseGetErrorString(status), status);                          \
    }                                                                          \
  }

namespace cuSPARSE {

template <typename DataType>
float CuSparseTest<DataType>::Multiply(const dCSR<DataType> &A,
                                       const dCSR<DataType> &B,
                                       dCSR<DataType> &c,
                                       uint32_t &cusparse_nnz) {
    float duration;
    int m, n, k;
    m = A.rows;
    n = B.cols;
    k = A.cols;
    c.reset();

    hipsparseHandle_t handle;
    hipsparseSpMatDescr_t matA;
    hipsparseSpMatDescr_t matB;
    hipsparseSpMatDescr_t matC;
    hipDeviceSynchronize();
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // ############################
    hipEventRecord(start);
    // ############################

    // Allocate memory for row indices
    hipMalloc(&(c.row_offsets), sizeof(uint32_t) * (A.rows + 1));

    // CUSPARSE APIs
    CHECK_CUSPARSE(hipsparseCreate(&handle))
    // Create sparse matrix A in CSR format
    CHECK_CUSPARSE(hipsparseCreateCsr(
        &matA, m, k, A.nnz, A.row_offsets, A.col_ids, A.data, HIPSPARSE_INDEX_32I,
        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F))
    CHECK_CUSPARSE(hipsparseCreateCsr(
        &matB, k, n, B.nnz, B.row_offsets, B.col_ids, B.data, HIPSPARSE_INDEX_32I,
        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F))
    CHECK_CUSPARSE(hipsparseCreateCsr(&matC, m, n, 0, c.row_offsets, NULL, NULL,
                                HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F))

    void *dBuffer1 = NULL, *dBuffer2 = NULL;
    size_t bufferSize1 = 0, bufferSize2 = 0;

    DataType alpha = 1.0;
    DataType beta = 0.0;
    hipsparseOperation_t opA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    hipsparseOperation_t opB = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    hipDataType computeType = HIP_R_32F;
    
    // SpGEMM Computation
    hipsparseSpGEMMDescr_t spgemmDesc;
    CHECK_CUSPARSE(hipsparseSpGEMM_createDescr(&spgemmDesc))
    
    // ask bufferSize1 bytes for external memory
    CHECK_CUSPARSE(hipsparseSpGEMM_workEstimation(
        handle, opA, opB, &alpha, matA, matB, &beta, matC, computeType,
        HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc, &bufferSize1, NULL))
    hipMalloc((void **)&dBuffer1, bufferSize1);
    
    // inspect the matrices A and B to understand the memory requirement for the
    // next step
    CHECK_CUSPARSE(hipsparseSpGEMM_workEstimation(
        handle, opA, opB, &alpha, matA, matB, &beta, matC, computeType,
        HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc, &bufferSize1, dBuffer1))
    
    // ask bufferSize2 bytes for external memory
    CHECK_CUSPARSE(hipsparseSpGEMM_compute(
        handle, opA, opB, &alpha, matA, matB, &beta, matC, computeType,
        HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc, &bufferSize2, NULL))
    hipMalloc((void **)&dBuffer2, bufferSize2);

    // compute the intermediate product of A * B
    CHECK_CUSPARSE(hipsparseSpGEMM_compute(
        handle, opA, opB, &alpha, matA, matB, &beta, matC, computeType,
        HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc, &bufferSize2, dBuffer2))
    
    // get matrix C non-zero entries C_nnz1
    int64_t cnrow, cncolmn, cnnz;
    CHECK_CUSPARSE(hipsparseSpMatGetSize(matC, &cnrow, &cncolmn, &cnnz))
    c.rows = cnrow;
    c.cols = cncolmn;
    c.nnz = cnnz;
    
    // allocate matrix C
    hipMalloc((void **)&c.col_ids, c.nnz * sizeof(uint32_t));
    hipMalloc((void **)&c.data, c.nnz * sizeof(DataType));
    cusparse_nnz = c.nnz;

    // update matC with the new pointers
    CHECK_CUSPARSE(
        hipsparseCsrSetPointers(matC, c.row_offsets, c.col_ids, c.data));

    // copy the final products to the matrix C
    CHECK_CUSPARSE(hipsparseSpGEMM_copy(handle, opA, opB, &alpha, matA, matB,
                                        &beta, matC, computeType,
                                        HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc));

    // destroy matrix/vector descriptors
    CHECK_CUSPARSE(hipsparseSpGEMM_destroyDescr(spgemmDesc));
    
    hipsparseDestroySpMat(matA);
    hipsparseDestroySpMat(matB);
    CHECK_CUSPARSE(hipsparseDestroySpMat(matC));
    hipsparseDestroy(handle);
    // ############################
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    // ############################

    hipEventElapsedTime(&duration, start, stop);
    hipFree(dBuffer1);
    hipFree(dBuffer2);
    hipDeviceSynchronize();
    return duration;
}

template float CuSparseTest<float>::Multiply(const dCSR<float> &A,
                                             const dCSR<float> &B,
                                             dCSR<float> &matOut,
                                             uint32_t &cusparse_nnz);
template float CuSparseTest<double>::Multiply(const dCSR<double> &A,
                                              const dCSR<double> &B,
                                              dCSR<double> &matOut,
                                              uint32_t &cusparse_nnz);

template <typename DataType>
void CuSparseTest<DataType>::Transpose(const dCSR<DataType> &A,
                                       dCSR<DataType> &AT) {}

} // namespace cuSPARSE